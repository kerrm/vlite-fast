#include "cuda_util.h"
#include <stdio.h>

void cudacheck (hipError_t err)
{
  if (err!=hipSuccess)
  {
    fprintf (stderr, "CUDA ERROR! %s\n",hipGetErrorString(err));
    //exit(0);
    throw 20;
  }
}

void cufftcheck(hipfftResult err){
  if (err!=HIPFFT_SUCCESS)
  {
    fprintf (stderr, "cufft ERROR! No. = %d\n", err);
    //exit(0);
    throw 20;
  }
}

void curandcheck(hiprandStatus_t err){
  if (err!=HIPRAND_STATUS_SUCCESS)
  {
    fprintf (stderr, "hiprand ERROR! No. = %d\n", err);
    //exit(0);
    throw 20;
  }
}
