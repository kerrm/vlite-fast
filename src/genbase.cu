#include "hip/hip_runtime.h"
/* Current status:

   11/14/2018

   Seems to be working fine, but I note that there are some issues with
   buffer pollution for the case of smeared pulse widths comparable to the
   buffer width.  NOT equal, I find empirically that for the data to look
   reasonable in a filterbank analysis the buffer should contain several
   periods worth of data.  This is currently checked for below.  It would
   be nice to understand why it isn't the more reasonable 
   buffer > dm_smearing but there are only so many hours in the day.

*/
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>
#include <string.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <time.h>
#include <hipfft/hipfft.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime_api.h>

#include "vdifio.h"

#include "dada_def.h"
#include "dada_hdu.h"
#include "ipcio.h"
#include "ascii_header.h"
#include "multilog.h"

#include "util.h"
#include "cuda_util.h"

#define DEVICE 0        //GPU. on furby: 0 = TITAN Black, 1 = GTX 780
#define NTHREAD 512
#define WRITE_DADA 0 // 0 write to file, 1 write to psrdada buffer
#define NMOMENT 256

#define VD_FRM 5032
#define VD_DAT 5000
#define VLITE_RATE 128000000
#define VLITE_FREQ 352.
#define VLITE_FRAME_RATE 25600


__global__ void init_dm_kernel (hipfftComplex *fker_dev, float dm, size_t n);
__global__ void set_profile (hipfftReal *fdat_dev, size_t current_sample, 
    size_t period, int skip_period, float ampl, size_t n);
__global__ void multiply_kernel (hipfftComplex* dat,hipfftComplex* ker, size_t n);
__global__ void swap_sideband (hipfftReal *dat, size_t n);
__global__ void setup_dstate (hiprandState *state);
__global__ void measure_moments (hipfftReal* dat, float *moments);
__global__ void add_rfi (hipfftReal *dat, size_t n, hiprandState *d_state,
    size_t current_sample, double tsamp_in_mus);
__global__ void digitize (float* idat, uint8_t* udat, size_t n);

void usage ()
{
  fprintf(stdout,"Usage: genbase [options]\n"
	  "-t seconds to simulate (default: 5)"
	  "-n observations to simulate (default: 1)\n"
	  "-p pulse period [s; default 0.5]"
    // this is used to simulate an FRB e.g. with -p 1.0 -k 100
	  "-k skip period [int; only produce a pulse every skip_period pulses]"
	  "-a amplitude as fraction of Tsys [default 0.05]"
	  "-s scale of second polarization relative to first [default 1.0]"
	  "-r seed for random number generator [long; default=42]"
	  "-d dm [default=30; NB this is about the largest feasible]"
	  "-e write to disk rather than psrdada buffer (default: false)"
	  "-f add RFI to observations (default: false)");
}

int main(int argc, char *argv[])
{
  double tobs = 5;
  double dm = 30;
  double pulse_period = 0.5;
  float ampls[2] = {0.05,0.05};
  float poln_ratio = 1.;
  int nobs = 1;
  int arg = 0;
  long seed = 42;
  int do_add_rfi = 0;
  int write_to_dada = 1;
  int skip_period = 1;
  while ((arg = getopt(argc, argv, "hfet:n:p:a:s:d:r:k:")) != -1) {

    switch (arg) {

    case 'h':
      usage ();
      return 0;
      
    case 't':
      if (sscanf (optarg, "%lf", &tobs) != 1) {
        fprintf (stderr, "genbase: could not read obs. time from %s\n", optarg);
        return -1;
      }
      break;

    case 'n':
      if (sscanf (optarg, "%d", &nobs) != 1) {
        fprintf (stderr, "writer: could not read num obs. from %s\n", optarg);
        return -1;
      }
      break;

    case 'a':
      if (sscanf (optarg, "%f", ampls) != 1) {
        fprintf (stderr, "genbase: could not read ampl.from %s\n", optarg);
        return -1;
      }
      break;

    case 's':
      if (sscanf (optarg, "%f", &poln_ratio) != 1) {
        fprintf (stderr, "genbase: could not pol'n ratio from %s\n", optarg);
        return -1;
      }
      break;

    case 'd':
      if (sscanf (optarg, "%lf", &dm) != 1) {
        fprintf (stderr, "genbase: could not read DM from %s\n", optarg);
        return -1;
      }
      break;

    case 'p':
      if (sscanf (optarg, "%lf", &pulse_period) != 1) {
        fprintf (stderr, "genbase: could not read period from %s\n", optarg);
        return -1;
      }
      break;

    case 'r':
      if (sscanf (optarg, "%li", &seed) != 1) {
        fprintf (stderr, "genbase: could not read seed from %s\n", optarg);
        return -1;
      }
      break;

    case 'f':
      printf ("genbase: adding RFI\n" );
      do_add_rfi = 1;
      break;

    case 'e':
      printf ("genbase: writing to disk\n" );
      write_to_dada = 0;
      break;


    case 'k':
      if (sscanf (optarg, "%d", &skip_period) != 1) {
        fprintf (stderr, "writer: could not read skip period from %s\n", optarg);
        return -1;
      }
      break;
    }
  }
  printf("Skip period =%d.\n",skip_period);

  // apply any polarization scaling
  ampls[1] = ampls[0] * poln_ratio;

  // set up sample counts for given DM
  double freq = 352;
  double freq_hi = 384;
  double freq_lo = 320;
  double tsamp = 1.0/VLITE_RATE; // NB real sampling
  printf("Sampling time is %g.\n",tsamp);
  double t_dm_lo = dm/2.41e-10*(1./(freq_lo*freq_lo)-1./(freq*freq)); // mus
  double t_dm_hi = dm/2.41e-10*(1./(freq*freq)-1./(freq_hi*freq_hi)); // mus
  printf ("DM smearing time to bottom of band is %.2f ms.\n",t_dm_lo*1e-3);
  printf ("DM smearing time to top of band is %.2f ms.\n",t_dm_hi*1e-3);
  printf ("deltaDM smearing time is %.2f ms.\n",(t_dm_lo-t_dm_hi)*1e-3);
  unsigned long n_dm_samp_lo = (unsigned long) t_dm_lo*1e-6/tsamp;
  unsigned long n_dm_samp_hi = (unsigned long) t_dm_hi*1e-6/tsamp;
  printf ("DM smearing samples to bottom of band is %li.\n",n_dm_samp_lo);
  printf ("DM smearing samples to top of band is %li.\n",n_dm_samp_hi);
  n_dm_samp_lo += (n_dm_samp_lo & 1); // make it even
  n_dm_samp_hi += (n_dm_samp_hi & 1); // make it even
  {
    unsigned long tmp = n_dm_samp_lo;
    n_dm_samp_lo = n_dm_samp_hi;
    n_dm_samp_hi = tmp;
  }
  unsigned long n_dm_samp = n_dm_samp_lo + n_dm_samp_hi;
  printf("DM total samples is %li.\n",n_dm_samp);

  // pulse properties
  size_t period_in_samples = size_t(pulse_period/tsamp);
  printf ("Pulse period is %li samples.\n",period_in_samples);

  // allocate memory for 1s of data; NB this is a large buffer, but because
  // of edge effects, will discard ~0.37s of data at DM=30!
  size_t buflen = VLITE_RATE/4;
  //size_t buflen = VLITE_RATE;

  if (buflen < 2*(n_dm_samp + period_in_samples))
  {
    fprintf (stderr, "Buffer not long enough to perform dedispersion!");
    exit (EXIT_FAILURE);
  }

  // initialize GPU properties
  cudacheck (hipSetDevice (DEVICE));
  int nsms;
  hipDeviceGetAttribute(&nsms,hipDeviceAttributeMultiprocessorCount,DEVICE);

  hipfftReal* fdat_dev; cudacheck (
  hipMalloc ((void**)&fdat_dev, sizeof(hipfftReal)*buflen) );

  // make a separate buffer to store the overlap; need one for each polarization
  hipfftReal* fovl_dev_p0; cudacheck (
  hipMalloc ((void**)&fovl_dev_p0, sizeof(hipfftReal)*n_dm_samp) );
  hipfftReal* fovl_dev_p1; cudacheck (
  hipMalloc ((void**)&fovl_dev_p1, sizeof(hipfftReal)*n_dm_samp) );
  hipfftReal* fovl_dev_pols[] = {fovl_dev_p0, fovl_dev_p1};

  // allocate memory for DM kernel
  hipfftComplex* fker_dev; cudacheck (
  hipMalloc ((void**)&fker_dev, sizeof(hipfftComplex)*(buflen/2+1)) );

  // allocate memory for FFT
  hipfftComplex* ffft_dev; cudacheck (
  hipMalloc ((void**)&ffft_dev, sizeof(hipfftComplex)*(buflen/2+1)) );

  // allocate memory for digitized output; only do the unpolluted samples
  size_t new_samps = buflen - n_dm_samp;
  uint8_t* udat_dev; cudacheck (
  hipMalloc ((void**)&udat_dev, new_samps) );

  // allocate memory and initialize state for hiprand; NB that we need a
  // state for *each* thread
  hiprandState *d_state;
  if (do_add_rfi)
  {
    hipMalloc(&d_state, sizeof(hiprandState)*NTHREAD*32);
    setup_dstate <<<32,NTHREAD>>> (d_state);
  }

  // allocate host memory; only copy over the unpolluted samples
  // leave room for a ragged edge VDIF frame
  size_t vdif_offset = 0;
  uint8_t* udat_host_p0; cudacheck (
  hipHostMalloc ((void**)&udat_host_p0, new_samps + VD_DAT) );
  uint8_t* udat_host_p1; cudacheck (
  hipHostMalloc ((void**)&udat_host_p1, new_samps + VD_DAT) );
  uint8_t* udat_host_pols[] = {udat_host_p0, udat_host_p1};

  // initialize DM kernel; also include correction for FFT normalization
  init_dm_kernel<<<32*nsms,NTHREAD>>> (fker_dev,dm,buflen/2+1);
  cudacheck ( hipGetLastError() );

  // set up RNG for voltage generated
  hiprandGenerator_t gen;
  curandcheck (hiprandCreateGenerator (&gen, HIPRAND_RNG_PSEUDO_DEFAULT) );
  curandcheck (hiprandSetPseudoRandomGeneratorSeed (gen, seed) );

  // set up the FFTs; NB same plan for forward and backward
  hipfftHandle plan_fwd,plan_bwd;
  checkCudaErrors (hipfftPlan1d (&plan_fwd,buflen,HIPFFT_R2C,1));
  checkCudaErrors (hipfftPlan1d (&plan_bwd,buflen,HIPFFT_C2R,1));

  // set up primary VDIF header
  char hdr_buff[32];
  vdif_header* hdr = (vdif_header*) hdr_buff;
  setVDIFBitsPerSample (hdr, 8);
  setVDIFFrameBytes (hdr, VD_FRM);
  setVDIFNumChannels (hdr,1);

  // if using PSRDADA, connect to the output buffer
  key_t key = 0x40;
  multilog_t* log=NULL;
  dada_hdu_t* hdu=NULL;
  FILE *output_fp=NULL;
  if (write_to_dada)
  {
    log = multilog_open ("genbase",0);
    hdu = dada_hdu_create (log);
    dada_hdu_set_key (hdu,key);
    dada_hdu_connect (hdu);
  }
  else
  {
    output_fp = myopen("/data/kerrm/baseband_sim.uw","wb");
  }


  // set time for current set of data; will change after generating apt.
  // no. of seconds
  for (int nseg=0; nseg < nobs; ++nseg) 
  {
  printf("Working on segment %d.\n",nseg);

  // initialize overlap buffer
  printf ("Setting up a buffer of %li with an overlap of %li.\n",buflen,n_dm_samp);
  printf ("Will lose %0.2f to edge effects.\n",double(n_dm_samp)/buflen);
  fflush (stdout) ;
  size_t current_sample = 0;

  //hipfftReal* new_start_p0 = fdat_dev_p0  + n_dm_samp;
  //hipfftReal* new_start_p1 = fdat_dev_p1  + n_dm_samp;
  for (int ipol=0; ipol < 2; ++ipol) {
    curandcheck (hiprandGenerateNormal (
        gen, (float*) fovl_dev_pols[ipol], n_dm_samp, 0, 1) );
    set_profile<<<32*nsms, NTHREAD>>> (

        fovl_dev_pols[ipol], current_sample, period_in_samples, 
        skip_period, 1+ampls[ipol], n_dm_samp);
    cudacheck ( hipGetLastError() );
  }
  current_sample += n_dm_samp;

  if (write_to_dada)
  {
    // connect to DADA buffer and set current system time for epoch
    dada_hdu_lock_write (hdu);
    setVDIFFrameTime (hdr, time (NULL) );

    // write psrdada output header values
    char* ascii_hdr = ipcbuf_get_next_write (hdu->header_block);
    dadacheck (ascii_header_set (ascii_hdr, "NAME", "%s", "B0833-45" ) );
    dadacheck (ascii_header_set (ascii_hdr, "NCHAN", "%d", 1) );
    dadacheck (ascii_header_set (ascii_hdr, "BANDWIDTH", "%lf", 64.0) );
    dadacheck (ascii_header_set (ascii_hdr, "CFREQ", "%lf", 352.0) );
    dadacheck (ascii_header_set (ascii_hdr, "NPOL", "%d", 2) );
    dadacheck (ascii_header_set (ascii_hdr, "NBIT", "%d", 8) );
    dadacheck (ascii_header_set (ascii_hdr, "RA", "%lf", 0.87180) );
    dadacheck (ascii_header_set (ascii_hdr, "DEC", "%lf", 0.72452) );
    // NB psrdada format has TSAMP in microseconds
    //dadacheck (ascii_header_set (ascii_hdr, "TSAMP", "%lf", tsamp*1e6) );
    // set up epoch appropriately -- first, make a tm struct for VDIF epoch
    struct tm tm_epoch = {0};
    int vdif_epoch = getVDIFEpoch (hdr);
    tm_epoch.tm_year = 100 + vdif_epoch/2;
    tm_epoch.tm_mon = 6*(vdif_epoch%2);
    time_t epoch_seconds = mktime (&tm_epoch) + getVDIFFrameEpochSecOffset (hdr);
    struct tm* utc_time = gmtime (&epoch_seconds);
    char dada_utc[64];
    strftime (dada_utc, 64, DADA_TIMESTR, utc_time);
    printf("UTC START: %s\n",dada_utc);
    dadacheck (ascii_header_set (ascii_hdr, "UTC_START", "%s", dada_utc) );
    printf("%s",ascii_hdr);
    ipcbuf_mark_filled (hdu->header_block, 4096);
  }

  double sec_to_sim = tobs;
  size_t end_sample = size_t(sec_to_sim/tsamp);
  size_t current_frame = 0;
  int frame_seconds = 0;

  // sanity checks on voltage levels
  //float moments[2] = {0,0};
  //float* moments_dev;
  //hipMalloc ((void**)&moments_dev, sizeof(float)*2);

  while (current_sample < end_sample)
  {
    
    for (int ipol=0; ipol < 2; ++ipol) {

      // copy overlap from previous input
      cudacheck (hipMemcpy (
          fdat_dev, fovl_dev_pols[ipol], 
          n_dm_samp*sizeof(hipfftReal), hipMemcpyDeviceToDevice) );

      // generate input to fill non-overlap region; generate real samps
      curandcheck (hiprandGenerateNormal (
          gen, (float*) fdat_dev+n_dm_samp, new_samps, 0, 1) );

      // set pulse profile
      set_profile<<<32*nsms, NTHREAD>>> (
          fdat_dev+n_dm_samp, current_sample, period_in_samples,
          skip_period,1.+ampls[ipol], new_samps);
      cudacheck ( hipGetLastError() );

      // copy input for next overlap to overlap buffer
      cudacheck (hipMemcpy (
            fovl_dev_pols[ipol], fdat_dev+buflen-n_dm_samp, 
          n_dm_samp*sizeof(hipfftReal), hipMemcpyDeviceToDevice) );

      // forward transform the input
      cufftcheck (hipfftExecR2C (plan_fwd, fdat_dev, ffft_dev) );

      // multiply by DM kernel
      multiply_kernel <<<32*nsms, NTHREAD>>> (ffft_dev, fker_dev, buflen/2+1);
      cudacheck ( hipGetLastError() );

      // inverse transform
      cufftcheck (hipfftExecC2R (plan_bwd, ffft_dev, fdat_dev) );

      // change to same sideband sense as VLITE
      swap_sideband <<<32*nsms, NTHREAD>>> (fdat_dev, buflen);
      cudacheck ( hipGetLastError() );

      /*
      // an optional sanity check on the moments; last calculation showed
      // they nicedly followed a standard normal distribution
      moments[0] = 0;
      moments[1] = 0;
      cudacheck (hipMemcpy (moments_dev, moments, 2*sizeof(float),
          hipMemcpyHostToDevice));

      measure_moments <<< buflen/256, 256>>> (fdat_dev, moments_dev);
      cudacheck ( hipGetLastError() );
      cudacheck (hipMemcpy (moments, moments_dev, 2*sizeof(float), hipMemcpyDeviceToHost) );
      moments[0] *= double(256)/buflen;
      moments[1] *= double(256)/buflen;
      printf ("moment2 %.6f %.6f\n", moments[0], sqrt(moments[0]));
      printf ("moment4 %.6f\n", moments[1]);
      */

      if (do_add_rfi)
      {
        // this sample offset means that RFI "phase" is referenced to the
        // very first sample written out; makes analysis easier
        add_rfi <<<32, NTHREAD>>> (fdat_dev, buflen, d_state, 
            current_sample-n_dm_samp-n_dm_samp_lo, tsamp*1e6);
        cudacheck ( hipGetLastError() );
      }

      // digitize to 8-bit uints; simultaneously select only valid samples
      digitize <<<32*nsms, NTHREAD>>> (
          (float*)(fdat_dev+n_dm_samp_lo), udat_dev, new_samps);
      cudacheck ( hipGetLastError() );

      // copy to host
      cudacheck (hipMemcpy (udat_host_pols[ipol] + vdif_offset, udat_dev, new_samps, hipMemcpyDeviceToHost) );
    } // end loop over polarizations

    current_sample += new_samps;

    // write to psrdada buffer or file
    size_t nframes = (new_samps + vdif_offset)/VD_DAT;

    if (write_to_dada)
    {
      for (size_t iframe = 0; iframe < nframes; ++iframe)
      {
        // update VDIF header
        if (current_frame == VLITE_FRAME_RATE)
        {
          frame_seconds ++;
          setVDIFFrameSecond (hdr, getVDIFFrameSecond (hdr) + 1);
          current_frame = 0;
        }
        setVDIFFrameNumber (hdr, current_frame);
        for (int ipol = 0; ipol < 2; ++ipol)
        {
          setVDIFThreadID(hdr, ipol);
          ipcio_write (hdu->data_block,hdr_buff,32);
          ipcio_write (hdu->data_block,(char*)(udat_host_pols[ipol] + VD_DAT*iframe), VD_DAT);
        }
        current_frame++;
      }
    }
    else
    {
      for (size_t iframe = 0; iframe < nframes; ++iframe)
      {
        // update VDIF header
        if (current_frame == VLITE_FRAME_RATE)
        {
          frame_seconds ++;
          setVDIFFrameSecond (hdr, getVDIFFrameSecond (hdr) + 1);
          current_frame = 0;
        }
        setVDIFFrameNumber (hdr, current_frame);
        for (int ipol = 0; ipol < 2; ++ipol)
        {
          setVDIFThreadID(hdr, ipol);
          fwrite(hdr_buff,1,32,output_fp);
          fwrite(udat_host_pols[ipol]+VD_DAT*iframe,1,VD_DAT,output_fp);
        }
        current_frame++;
      }
    }

    // copy remainder to beginning of buffer for next time
    // NB this is always a number <5032 samples, i.e. one frame
    // TODO -- figure this out and what to do RE polarization
    // MTK -- obviously I don't now know what the above TODO means
    size_t tocopy = new_samps + vdif_offset - nframes*VD_DAT;
    if (tocopy > 0)
    {
      cudacheck (hipMemcpy (udat_host_p0, udat_host_p0 + nframes*VD_DAT, tocopy, hipMemcpyHostToHost) );
      cudacheck (hipMemcpy (udat_host_p1, udat_host_p1 + nframes*VD_DAT, tocopy, hipMemcpyHostToHost) );
      vdif_offset = tocopy;
    }
   
  } // end loop over samples

  if (hdu)
    dada_hdu_unlock_write (hdu);
  if (output_fp)
    fclose (output_fp);

  } // end loop over observations

  // this cleanup a bit trivial at end of program
  hiprandDestroyGenerator (gen);
  hipFree (fdat_dev);
  hipFree (fovl_dev_p0);
  hipFree (fovl_dev_p1);
  hipFree (fker_dev);
  hipFree (ffft_dev);
  hipFree (udat_dev);
  hipHostFree (udat_host_p1);
  hipHostFree (udat_host_p0);

}



// set up DM kernel
__global__ void init_dm_kernel(hipfftComplex *ker, float dm, size_t n)
{
  // i is the index into the array, and the FFT is arranged with frequencies
  // 0/N, 1/N, ...  (real to complex FFT, only +ve freqs)
  for (
      int i = threadIdx.x + blockIdx.x*blockDim.x; 
      i < n; 
      i += blockDim.x*gridDim.x)
  {
    // NB hardcoded bw and freq for now
    double freq = (64.*double(i))/double(n);
    double freq0 = 320.;
    double arg = (2*M_PI*dm/2.41e-10)*freq*freq/(freq0*freq0*(freq0+freq));
    double rcos,rsin;
    sincos(arg, &rsin, &rcos);
    ker[i].x = rcos/(2*(n-1));
    ker[i].y = rsin/(2*(n-1));

    // make a slightly more realistic bandpass; this has a relatively fast,
    // asymmetric taper on each side as well as a modest ramp
    freq *= 1./64;
    double scale = 1-exp(-(freq*freq)/(0.05*0.05));
    scale -= exp(-((1-freq)*(1-freq))/(0.10*0.10));
    scale *= (1+0.20*freq);
    ker[i].x *= scale;
    ker[i].y *= scale;
  }
}

__global__ void set_profile(hipfftReal *dat, size_t current_sample, 
        size_t period_in_samples, int skip_period, float ampl, size_t n)
{
  for (
      int i = threadIdx.x + blockIdx.x*blockDim.x; 
      i < n; 
      i += blockDim.x*gridDim.x)
  {
    size_t sample = current_sample + i;
    /*
    // to emulate complex mixing, every odd sample should be multiplied by
    // -i, which is specific to the centre frequency / bandwidth of this app
    if (sample & 1)
    {
      float datx = dat[i].x;
      dat[i].x = dat[i].y;
      dat[i].y = -datx;
    }
    */
    // calculate integer phase for use in skip period
    int phasei = sample/period_in_samples;
    float phasef = float(sample-phasei*period_in_samples)/period_in_samples;
    if ((phasef < 0.03) && ((phasei%skip_period)==0))
    //if (phasef < 0.03)
    {
      //float tmp =1-abs(phase/0.025-1);
      //float amp = 1+tmp*tmp;
      //dat[i] *= amp;
      dat[i] *= ampl;
    }
  }
}

__global__ void multiply_kernel(hipfftComplex* dat,hipfftComplex* ker, size_t n)
{
  for (
      int i = threadIdx.x + blockIdx.x*blockDim.x; 
      i < n; 
      i += blockDim.x*gridDim.x)
  {
    float datx = dat[i].x;
    dat[i].x = datx*ker[i].x - dat[i].y*ker[i].y;
    dat[i].y = datx*ker[i].y + dat[i].y*ker[i].x;
  }
}

__global__ void measure_moments (hipfftReal* dat, float *moments)
{
  unsigned int tid = threadIdx.x;
  size_t offset = blockIdx.x*NMOMENT;

  // general plan of work: do explicit sum within each warp
  volatile __shared__ float data2[256];
  volatile __shared__ float data4[256];

  data2[tid] = dat[offset + tid]*dat[offset + tid];
  data4[tid] = data2[tid]*data2[tid];

  __syncthreads ();

  if (tid < 128)
  {
    data2[tid] += data2[tid + 128];
    data4[tid] += data4[tid + 128];
  }
  __syncthreads ();

  if (tid < 64)
  {
    data2[tid] += data2[tid + 64];
    data4[tid] += data4[tid + 64];
  }
  __syncthreads ();

  if (tid < 32)
  {
    data2[tid] += data2[tid + 32];
    data4[tid] += data4[tid + 32];
    data2[tid] += data2[tid + 16];
    data4[tid] += data4[tid + 16];
    data2[tid] += data2[tid + 8];
    data4[tid] += data4[tid + 8];
    data2[tid] += data2[tid + 4];
    data4[tid] += data4[tid + 4];
    data2[tid] += data2[tid + 2];
    data4[tid] += data4[tid + 2];
  }

  if (tid==0)
  {
    data2[tid] += data2[tid + 1];
    data4[tid] += data4[tid + 1];
    atomicAdd (moments + 0, data2[0] / NMOMENT );
    atomicAdd (moments + 1, data4[0] / NMOMENT );
  }
}

__global__ void swap_sideband(hipfftReal* dat, size_t n)
{
  for (
      int i = threadIdx.x + blockIdx.x*blockDim.x; 
      i < n; 
      i += blockDim.x*gridDim.x)
  {
    if (i & 1)
      dat[i] = -dat[i];
  }
}

__global__ void setup_dstate(hiprandState *state)
{

  int idx = threadIdx.x+blockDim.x*blockIdx.x;
  hiprand_init(idx+1233456, 0, 0, &state[idx]);
}

// Add roughly 1 mus of RFI every 10 mus of data.
__global__ void add_rfi(hipfftReal* dat, size_t n, hiprandState *d_state, size_t current_sample, double tsamp_in_mus)
{
  hiprandState *state = &d_state[threadIdx.x + blockIdx.x*NTHREAD];
  for (
      int i = threadIdx.x + blockIdx.x*blockDim.x; 
      i < n; 
      i += blockDim.x*gridDim.x)
  {

    float phase = fmodf((i+current_sample) * (tsamp_in_mus/11.3),1);
    if (phase < 0.1)
    {
      // Quick and dirty, add a random uniform signal
      dat[i] += 5.*(hiprand_uniform (state) - 0.5);
    }
  }
}

__global__ void digitize(float* idat, uint8_t* udat, size_t n)
{
  for (
      int i = threadIdx.x + blockIdx.x*blockDim.x; 
      i < n; 
      i += blockDim.x*gridDim.x)
  {
    // add an extra 2 here for overhead in case we make it bright
    //float tmp = idat[i]/0.02957/2 + 127.5;
    // this normalization appears to be more consistent with the VLITE
    // digitizers, which have a mean of 128
    float tmp = idat[i]/0.02957/2 + 128.5;
    if (tmp <= 0)
      udat[i] = 0;
    else if (tmp >= 255)
      udat[i] = 255;
    else
      udat[i] = (uint8_t) tmp;
  }
}

