#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>
#include <string.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <time.h>
#include <hipfft/hipfft.h>
#include <hiprand.h>
#include <hip/hip_runtime_api.h>

#include "vdifio.h"

#include "dada_def.h"
#include "dada_hdu.h"
#include "ipcio.h"
#include "ascii_header.h"
#include "multilog.h"

#include "util.h"
#include "cuda_util.h"

#define DEVICE 0        //GPU. on furby: 0 = TITAN Black, 1 = GTX 780
#define NTHREAD 512
#define WRITE_DADA 1 // write to psrdada output

#define VD_FRM 5032
#define VD_DAT 5000
#define VLITE_RATE 128000000
#define VLITE_FREQ 352.
#define VLITE_FRAME_RATE 25600


__global__ void init_dm_kernel(hipfftComplex *fker_dev, float dm, size_t n);
__global__ void set_profile(hipfftReal *fdat_dev, size_t current_sample, 
    size_t period, float ampl, size_t n);
__global__ void multiply_kernel(hipfftComplex* dat,hipfftComplex* ker, size_t n);
__global__ void swap_sideband(hipfftReal *dat, size_t n);
__global__ void digitize(float* idat, uint8_t* udat, size_t n);

void usage ()
{
  fprintf(stdout,"Usage: genbase [options]\n"
	  "-t seconds to simulate (default: 5)"
	  "-p pulse period [s; default 0.5]"
	  "-a amplitude as fraction of Tsys [default 0.05]"
	  "-s scale of second polarization relative to first [default 1.0]"
	  "-r seed for random number generator [long; default=42]"
	  "-d dm [default=30; NB this is about the largest feasible]"
	  "-n observations to simulate (default: 1)\n");
}

int main(int argc, char *argv[])
{
  double tobs = 5;
  double dm = 30;
  double pulse_period = 0.5;
  float ampls[2] = {1.05,1.05};
  int nobs = 1;
  int arg = 0;
  long seed = 42;
  while ((arg = getopt(argc, argv, "ht:n:p:a:s:d:r:")) != -1) {

    switch (arg) {

    case 'h':
      usage ();
      return 0;
      
    case 't':
      if (sscanf (optarg, "%lf", &tobs) != 1) {
        fprintf (stderr, "genbase: could not read obs. time from %s\n", optarg);
        return -1;
      }
      break;

    case 'n':
      if (sscanf (optarg, "%d", &nobs) != 1) {
        fprintf (stderr, "writer: could not read num obs. from %s\n", optarg);
        return -1;
      }
      break;

    case 'a':
      if (sscanf (optarg, "%f", ampls) != 1) {
        fprintf (stderr, "genbase: could not read ampl.from %s\n", optarg);
        return -1;
      }

    case 's':
      float scale;
      if (sscanf (optarg, "%f", &scale) != 1) {
        fprintf (stderr, "genbase: could not read scale from %s\n", optarg);
        return -1;
      }
      ampls[1] = ampls[0]*scale;
      break;

    case 'd':
      if (sscanf (optarg, "%lf", &dm) != 1) {
        fprintf (stderr, "genbase: could not read DM from %s\n", optarg);
        return -1;
      }

    case 'p':
      if (sscanf (optarg, "%lf", &pulse_period) != 1) {
        fprintf (stderr, "genbase: could not read period from %s\n", optarg);
        return -1;
      }

    case 'r':
      if (sscanf (optarg, "%li", &seed) != 1) {
        fprintf (stderr, "genbase: could not read seed from %s\n", optarg);
        return -1;
      }

    }
  }

  // set up sample counts for given DM
  double freq = 352;
  double freq_hi = 384;
  double freq_lo = 320;
  double tsamp = 1.0/VLITE_RATE; // NB real sampling
  printf("Sampling time is %g.\n",tsamp);
  double t_dm_lo = dm/2.41e-10*(1./(freq_lo*freq_lo)-1./(freq*freq)); // mus
  double t_dm_hi = dm/2.41e-10*(1./(freq*freq)-1./(freq_hi*freq_hi)); // mus
  printf ("DM smearing time to bottom of band is %2f.\n",t_dm_lo);
  printf ("DM smearing time to top of band is %.2f.\n",t_dm_hi);
  unsigned long n_dm_samp_lo = (unsigned long) t_dm_lo*1e-6/tsamp;
  unsigned long n_dm_samp_hi = (unsigned long) t_dm_hi*1e-6/tsamp;
  printf ("DM smearing samples to bottom of band is %li.\n",n_dm_samp_lo);
  printf ("DM smearing samples to top of band is %li.\n",n_dm_samp_hi);
  n_dm_samp_lo += (n_dm_samp_lo & 1); // make it even
  n_dm_samp_hi += (n_dm_samp_hi & 1); // make it even
  unsigned long n_dm_samp = n_dm_samp_lo + n_dm_samp_hi;
  printf("DM total samples is %li.\n",n_dm_samp);

  // initialize GPU properties
  cudacheck (hipSetDevice (DEVICE));
  int nsms;
  hipDeviceGetAttribute(&nsms,hipDeviceAttributeMultiprocessorCount,DEVICE);

  // allocate memory for 1s of data; NB this is a large buffer, but because
  // of edge effects, will discard ~0.37s of data at DM=30!
  size_t buflen = VLITE_RATE/2;
  hipfftReal* fdat_dev; cudacheck (
  hipMalloc ((void**)&fdat_dev, sizeof(hipfftReal)*buflen) );

  // make a separate buffer to store the overlap; need one for each polarization
  hipfftReal* fovl_dev_p0; cudacheck (
  hipMalloc ((void**)&fovl_dev_p0, sizeof(hipfftReal)*n_dm_samp) );
  hipfftReal* fovl_dev_p1; cudacheck (
  hipMalloc ((void**)&fovl_dev_p1, sizeof(hipfftReal)*n_dm_samp) );
  hipfftReal* fovl_dev_pols[] = {fovl_dev_p0, fovl_dev_p1};

  // allocate memory for DM kernel
  hipfftComplex* fker_dev; cudacheck (
  hipMalloc ((void**)&fker_dev, sizeof(hipfftComplex)*(buflen/2+1)) );

  // allocate memory for FFT
  hipfftComplex* ffft_dev; cudacheck (
  hipMalloc ((void**)&ffft_dev, sizeof(hipfftComplex)*(buflen/2+1)) );

  // allocate memory for digitized output; only do the unpolluted samples
  size_t new_samps = buflen - n_dm_samp;
  uint8_t* udat_dev; cudacheck (
  hipMalloc ((void**)&udat_dev, new_samps) );

  // allocate host memory; only copy over the unpolluted samples
  // leave room for a ragged edge VDIF frame
  size_t vdif_offset = 0;
  uint8_t* udat_host_p0; cudacheck (
  hipHostMalloc ((void**)&udat_host_p0, new_samps + VD_DAT) );
  uint8_t* udat_host_p1; cudacheck (
  hipHostMalloc ((void**)&udat_host_p1, new_samps + VD_DAT) );
  uint8_t* udat_host_pols[] = {udat_host_p0, udat_host_p1};

  // initialize DM kernel; also include correction for FFT normalization
  init_dm_kernel<<<32*nsms,NTHREAD>>> (fker_dev,dm,buflen/2+1);
  cudacheck ( hipGetLastError() );

  // set up RNG for voltage generated
  hiprandGenerator_t gen;
  curandcheck (hiprandCreateGenerator (&gen, HIPRAND_RNG_PSEUDO_DEFAULT) );
  curandcheck (hiprandSetPseudoRandomGeneratorSeed (gen, seed) );

  // set up the FFTs; NB same plan for forward and backward
  hipfftHandle plan_fwd,plan_bwd;
  checkCudaErrors (hipfftPlan1d (&plan_fwd,buflen,HIPFFT_R2C,1));
  checkCudaErrors (hipfftPlan1d (&plan_bwd,buflen,HIPFFT_C2R,1));

  // set up primary VDIF header
  char hdr_buff[32];
  vdif_header* hdr = (vdif_header*) hdr_buff;
  setVDIFBitsPerSample (hdr, 8);
  setVDIFFrameBytes (hdr, VD_FRM);
  setVDIFNumChannels (hdr,1);

  // connect to the output buffer
#if WRITE_DADA
  key_t key = 0x40;
  multilog_t* log = multilog_open ("genbase",0);
  dada_hdu_t* hdu = dada_hdu_create (log);
  dada_hdu_set_key (hdu,key);
  dada_hdu_connect (hdu);
#else
  FILE *output_fp = myopen("/data/VLITE/kerrm/baseband_sim.uw","wb");
#endif

  // pulse properties
  size_t period = size_t(pulse_period/tsamp);
  printf ("Pulse period is %li samples.\n",period);


  // set time for current set of data; will change after generating apt.
  // no. of seconds
  for (int nseg=0; nseg < nobs; ++nseg) 
  {
  printf("Working on segment %d.\n",nseg);

  // initialize overlap buffer
  printf ("Setting up a buffer of %li with an overlap of %li.\n",buflen,n_dm_samp);
  printf ("Will lose %0.2f to edge effects.\n",double(n_dm_samp)/buflen);
  fflush (stdout) ;
  size_t current_sample = 0;

  //hipfftReal* new_start_p0 = fdat_dev_p0  + n_dm_samp;
  //hipfftReal* new_start_p1 = fdat_dev_p1  + n_dm_samp;
  for (int ipol=0; ipol < 2; ++ipol) {
    curandcheck (hiprandGenerateNormal (
        gen, (float*) fovl_dev_pols[ipol], n_dm_samp, 0, 1) );
    set_profile<<<32*nsms, NTHREAD>>> (
        fovl_dev_pols[ipol], current_sample, period, 1+ampls[ipol], n_dm_samp);
    cudacheck ( hipGetLastError() );
  }
  current_sample += n_dm_samp;

  // connect to DADA buffer and set current system time for epoch
  dada_hdu_lock_write (hdu);
  setVDIFFrameTime (hdr, time (NULL) );

  // write psrdada output header values
  char* ascii_hdr = ipcbuf_get_next_write (hdu->header_block);
  dadacheck (ascii_header_set (ascii_hdr, "NAME", "%s", "B0833-45" ) );
  dadacheck (ascii_header_set (ascii_hdr, "NCHAN", "%d", 1) );
  dadacheck (ascii_header_set (ascii_hdr, "BANDWIDTH", "%lf", 64) );
  dadacheck (ascii_header_set (ascii_hdr, "CFREQ", "%lf", 352) );
  dadacheck (ascii_header_set (ascii_hdr, "NPOL", "%d", 2) );
  dadacheck (ascii_header_set (ascii_hdr, "NBIT", "%d", 8) );
  dadacheck (ascii_header_set (ascii_hdr, "RA", "%lf", 0.87180) );
  dadacheck (ascii_header_set (ascii_hdr, "DEC", "%lf", 0.72452) );
  // NB psrdada format has TSAMP in microseconds
  //dadacheck (ascii_header_set (ascii_hdr, "TSAMP", "%lf", tsamp*1e6) );
  // set up epoch appropriately -- first, make a tm struct for VDIF epoch
  struct tm tm_epoch = {0};
  int vdif_epoch = getVDIFEpoch (hdr);
  tm_epoch.tm_year = 100 + vdif_epoch/2;
  tm_epoch.tm_mon = 6*(vdif_epoch%2);
  time_t epoch_seconds = mktime (&tm_epoch) + getVDIFFrameEpochSecOffset (hdr);
  struct tm* utc_time = gmtime (&epoch_seconds);
  char dada_utc[64];
  strftime (dada_utc, 64, DADA_TIMESTR, utc_time);
  printf("UTC START: %s\n",dada_utc);
  dadacheck (ascii_header_set (ascii_hdr, "UTC_START", "%s", dada_utc) );
  printf("%s",ascii_hdr);
  ipcbuf_mark_filled (hdu->header_block, 4096);

  double sec_to_sim = tobs;
  size_t end_sample = size_t(sec_to_sim/tsamp);
  size_t current_frame = 0;
  int frame_seconds = 0;

  while (current_sample < end_sample)
  {
    
    for (int ipol=0; ipol < 2; ++ipol) {

      // copy overlap from previous input
      cudacheck (hipMemcpy (
          fdat_dev, fovl_dev_pols[ipol], 
          n_dm_samp*sizeof(hipfftReal), hipMemcpyDeviceToDevice) );

      // generate input to fill non-overlap region; generate real samps
      curandcheck (hiprandGenerateNormal (
          gen, (float*) fdat_dev+n_dm_samp, new_samps, 0, 1) );

      // set pulse profile
      set_profile<<<32*nsms, NTHREAD>>> (
          fdat_dev+n_dm_samp, current_sample, period, 1.+ampls[ipol], new_samps);
      cudacheck ( hipGetLastError() );

      // copy input for next overlap to overlap buffer
      cudacheck (hipMemcpy (fovl_dev_pols[ipol], fdat_dev+buflen-n_dm_samp, 
          n_dm_samp*sizeof(hipfftReal), hipMemcpyDeviceToDevice) );

      // forward transform the input
      cufftcheck (hipfftExecR2C (plan_fwd, fdat_dev, ffft_dev) );

      // multiply by DM kernel
      multiply_kernel <<<32*nsms, NTHREAD>>> (ffft_dev, fker_dev, buflen/2+1);
      cudacheck ( hipGetLastError() );

      // inverse transform
      cufftcheck (hipfftExecC2R (plan_bwd, ffft_dev, fdat_dev) );

      // change to same sideband sense as VLITE
      swap_sideband <<<32*nsms, NTHREAD>>> (fdat_dev, buflen);
      cudacheck ( hipGetLastError() );

      // digitize to 8-bit uints; while doing this, select only valid samples
      digitize <<<32*nsms, NTHREAD>>> (
          (float*)(fdat_dev+n_dm_samp_lo), udat_dev, new_samps);
      cudacheck ( hipGetLastError() );

      // copy to host
      cudacheck (hipMemcpy (udat_host_pols[ipol] + vdif_offset, udat_dev, new_samps, hipMemcpyDeviceToHost) );
    }

    current_sample += new_samps;

    // write to psrdada buffer
    size_t nframes = (new_samps + vdif_offset)/VD_DAT;
    //printf("Will write out %d frames.\n",nframes);
    for (size_t iframe = 0; iframe < nframes; ++iframe)
    {
      // update VDIF header
      if (current_frame == VLITE_FRAME_RATE)
      {
        frame_seconds ++;
        setVDIFFrameSecond (hdr, getVDIFFrameSecond (hdr) + 1);
        current_frame = 0;
      }
      setVDIFFrameNumber (hdr, current_frame);
      for (int ipol = 0; ipol < 2; ++ipol)
      {
        setVDIFThreadID(hdr, ipol);
#if WRITE_DADA
        ipcio_write (hdu->data_block,hdr_buff,32);
        ipcio_write (hdu->data_block,(char*)(udat_host_pols[ipol] + VD_DAT*iframe), VD_DAT);
#else
        fwrite(hdr_buff,1,32,output_fp);
        fwrite(udat_host_pols[ipol]+VD_DAT*iframe,1,VD_DAT,output_fp);
#endif
      }
      current_frame++;
    }

    // copy remainder to beginning of buffer for next time
    // TODO -- figure this out and what to do RE polarization
    size_t tocopy = new_samps + vdif_offset - nframes*VD_DAT;
    if (tocopy > 0)
    {
      cudacheck (hipMemcpy (udat_host_p0, udat_host_p0 + nframes*VD_DAT, tocopy, hipMemcpyHostToHost) );
      cudacheck (hipMemcpy (udat_host_p1, udat_host_p1 + nframes*VD_DAT, tocopy, hipMemcpyHostToHost) );
      vdif_offset = tocopy;
    }
   
  }
  //printf("Advanced the frame second by %d.\n",frame_seconds);

#if WRITE_DADA
  dada_hdu_unlock_write (hdu);
#else
  fclose (output_fp);
#endif
  }

  // this cleanup a bit trivial at end of program
  hiprandDestroyGenerator (gen);
  hipFree (fdat_dev);
  hipFree (fovl_dev_p0);
  hipFree (fovl_dev_p1);
  hipFree (fker_dev);
  hipFree (ffft_dev);
  hipFree (udat_dev);
  hipHostFree (udat_host_p1);
  hipHostFree (udat_host_p0);

}



// set up DM kernel
__global__ void init_dm_kernel(hipfftComplex *ker, float dm, size_t n)
{
  // i is the index into the array, and the FFT is arranged with frequencies
  // 0/N, 1/N, ...  (real to complex FFT, only +ve freqs)
  for (
      int i = threadIdx.x + blockIdx.x*blockDim.x; 
      i < n; 
      i += blockDim.x*gridDim.x)
  {
    // NB hardcoded bw and freq for now
    double freq = (64.*double(i))/double(n);
    double freq0 = 320.;
    double arg = (2*M_PI*dm/2.41e-10)*freq*freq/(freq0*freq0*(freq0+freq));
    double rcos,rsin;
    sincos(arg, &rsin, &rcos);
    ker[i].x = rcos/(2*(n-1));
    ker[i].y = rsin/(2*(n-1));
  }
}

__global__ void set_profile(hipfftReal *dat, size_t current_sample, 
        size_t period, float ampl, size_t n)
{
  for (
      int i = threadIdx.x + blockIdx.x*blockDim.x; 
      i < n; 
      i += blockDim.x*gridDim.x)
  {
    size_t sample = current_sample + i;
    /*
    // to emulate complex mixing, every odd sample should be multiplied by
    // -i, which is specific to the centre frequency / bandwidth of this app
    if (sample & 1)
    {
      float datx = dat[i].x;
      dat[i].x = dat[i].y;
      dat[i].y = -datx;
    }
    */
    sample -= (sample/period)*period;
    float phase = float(sample)/period;
    if (phase < 0.05)
    {
      //float tmp =1-abs(phase/0.025-1);
      //float amp = 1+tmp*tmp;
      //dat[i] *= amp;
      dat[i] *= ampl;
    }
  }
}

__global__ void multiply_kernel(hipfftComplex* dat,hipfftComplex* ker, size_t n)
{
  for (
      int i = threadIdx.x + blockIdx.x*blockDim.x; 
      i < n; 
      i += blockDim.x*gridDim.x)
  {
    float datx = dat[i].x;
    dat[i].x = datx*ker[i].x - dat[i].y*ker[i].y;
    dat[i].y = datx*ker[i].y + dat[i].y*ker[i].x;
  }
}

__global__ void swap_sideband(hipfftReal* dat, size_t n)
{
  for (
      int i = threadIdx.x + blockIdx.x*blockDim.x; 
      i < n; 
      i += blockDim.x*gridDim.x)
  {
    if (i & 1)
      dat[i] = -dat[i];
  }
}

__global__ void digitize(float* idat, uint8_t* udat, size_t n)
{
  for (
      int i = threadIdx.x + blockIdx.x*blockDim.x; 
      i < n; 
      i += blockDim.x*gridDim.x)
  {
    // add an extra 2 here for overhead in case we make it bright
    //float tmp = idat[i]/0.02957/2 + 127.5;
    // this normalization appears to be more consistent with the VLITE
    // digitizers, which have a mean of 128
    float tmp = idat[i]/0.02957/2 + 128.5;
    if (tmp <= 0)
      udat[i] = 0;
    else if (tmp >= 255)
      udat[i] = 255;
    else
      udat[i] = (uint8_t) tmp;
  }
}

